#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

const int INF = 60001;
void input(char *inFileName);
void output(char *outFileName);
void block_FW(int B);
__global__ void phase1(int B, int r, int *device_Dist, int tn);
__global__ void phase2(int B, int r, int *device_Dist, int tn);
__global__ void phase3(int B, int r, int *device_Dist, int tn);
/* Get ceil(a / b) */
__device__ __host__ int ceil(int a, int b) {
    return (a + b - 1) / b;
}

int n, m, tn;
int *Dist, *device_Dist;
const int B = 64;

int main(int argc, char* argv[]){
    input(argv[1]);
    block_FW(B);
    output(argv[2]);
    return 0;
}

void input(char *inFileName){
    FILE *file = fopen(inFileName, "rb");
    fread(&n, sizeof(int), 1, file);
    fread(&m, sizeof(int), 1, file);

    tn = ceil(n, 64) * 64;
    hipHostMalloc(&Dist, tn*tn*sizeof(int), hipHostMallocDefault);
    for(int i = 0; i < tn; i++){
        for(int j = 0; j < tn; j++){
            Dist[i*tn+j] = (i==j&&i<n)?0:INF;
        }
    }

    int pair[3];
    for(int i = 0; i < m; i++){
        fread(pair, sizeof(int), 3, file);
        Dist[pair[0]*tn+pair[1]] = pair[2];
    }
    fclose(file);
}

void output(char *outFileName){
    FILE *file = fopen(outFileName, "w");
    for(int i = 0; i < n; i++){
        fwrite(&Dist[i*tn], sizeof(int), n, file);
    }
    fclose(file);
    hipHostFree(Dist);
}

void block_FW(int B){
    hipMalloc(&device_Dist, tn*tn*sizeof(int));
    hipMemcpy(device_Dist, Dist, tn*tn*sizeof(int), hipMemcpyHostToDevice);
    int round = tn/64;
    dim3 num_thds(32, 32);
    dim3 num_blks_ph2(2, round-1);
    dim3 num_blks_ph3(round-1, round-1);
    for(int r = 0; r < round; r++){
        phase1 <<<1, num_thds>>> (B, r, device_Dist, tn);
        phase2 <<<num_blks_ph2, num_thds>>> (B, r, device_Dist, tn);
        phase3 <<<num_blks_ph3, num_thds>>> (B, r, device_Dist, tn);
    }
    hipMemcpy(Dist, device_Dist, tn*tn*sizeof(int), hipMemcpyDeviceToHost);
    hipFree(device_Dist);
}

__global__ void phase1(int B, int r, int *device_Dist, int tn){
    __shared__ int s[64*64];
    int blk_i = r<<6, blk_j = r<<6;
    int i = threadIdx.y, j = threadIdx.x;

    #pragma unroll
    for(int x = 0; x < 2; x++){
        #pragma unroll
        for(int y = 0; y < 2; y++){
            s[(i+32 * y)*64+(j+32 * x)] = device_Dist[(blk_i+(i+32 * y))*tn+(blk_j+(j+32 * x))];
        }
    }
    __syncthreads();

    #pragma unroll 48
    for(int k = 0; k < 64; k++){
        #pragma unroll
        for(int x = 0; x < 2; x++){
            #pragma unroll
            for(int y = 0; y < 2; y++){
                s[(i+32 * y)*64+(j+32 * x)] = min(s[(i+32 * y)*64+(j+32 * x)], s[(i+32 * y)*64+k]+s[k*64+(j+32 * x)]);
            }
        }
        __syncthreads();
    }

    #pragma unroll
    for(int x = 0; x < 2; x++){
        #pragma unroll
        for(int y = 0; y < 2; y++){
            device_Dist[(blk_i+(i+32 * y))*tn+(blk_j+(j+32 * x))] = s[(i+32 * y)*64+(j+32 * x)];
        }
    }
}

__global__ void phase2(int B, int r, int *device_Dist, int tn){
    __shared__ int s[2*64*64];
    int blk_i = (blockIdx.x*r+(!blockIdx.x)*(blockIdx.y+(blockIdx.y>=r)))<<6;
    int blk_j = (blockIdx.x*(blockIdx.y+(blockIdx.y>=r))+(!blockIdx.x)*r)<<6;
    int blk_p = r<<6;
    int i = threadIdx.y, j = threadIdx.x;

    int val0 = device_Dist[(blk_i+i)*tn+(blk_j+j)];
    int val1 = device_Dist[(blk_i+i)*tn+(blk_j+(j+32))];
    int val2 = device_Dist[(blk_i+(i+32))*tn+(blk_j+j)];
    int val3 = device_Dist[(blk_i+(i+32))*tn+(blk_j+(j+32))];


    #pragma unroll
    for(int x = 0; x < 2; x++){
        #pragma unroll
        for(int y = 0; y < 2; y++){
            s[(i+32*y)*64+(j+32*x)] = device_Dist[(blk_i+(i+32*y))*tn+(blk_p+(j+32*x))];
            s[4096+(i+32*y)*64+(j+32*x)] = device_Dist[(blk_p+(i+32*y))*tn+(blk_j+(j+32*x))];
        }
    }
    __syncthreads();
    #pragma unroll 48
    for(int k = 0; k < 64; k++){
        val0 = min(val0, s[i*64+k]+s[4096+k*64+j]);
        val1 = min(val1, s[i*64+k]+s[4096+k*64+(j+32)]);
        val2 = min(val2, s[(i+32)*64+k]+s[4096+k*64+j]);
        val3 = min(val3, s[(i+32)*64+k]+s[4096+k*64+(j+32)]);
    }

    device_Dist[(blk_i+i)*tn+(blk_j+j)] = val0;
    device_Dist[(blk_i+i)*tn+(blk_j+(j+32))] = val1;
    device_Dist[(blk_i+(i+32))*tn+(blk_j+j)] = val2;
    device_Dist[(blk_i+(i+32))*tn+(blk_j+(j+32))] = val3;
}

__global__ void phase3(int B, int r, int *device_Dist, int tn){
    __shared__ int s[2*64*64];
    int blk_i = (blockIdx.x+(blockIdx.x>=r))<<6, blk_j = (blockIdx.y+(blockIdx.y>=r))<<6, blk_p = r<<6;
    int i = threadIdx.y, j = threadIdx.x;

    int val0 = device_Dist[(blk_i+i)*tn+(blk_j+j)];
    int val1 = device_Dist[(blk_i+i)*tn+(blk_j+(j+32))];
    int val2 = device_Dist[(blk_i+(i+32))*tn+(blk_j+j)];
    int val3 = device_Dist[(blk_i+(i+32))*tn+(blk_j+(j+32))];

    #pragma unroll
    for(int x = 0; x < 2; x++){
        #pragma unroll
        for(int y = 0; y < 2; y++){
            s[(i+32*y)*64+(j+32*x)] = device_Dist[(blk_i+(i+32*y))*tn+(blk_p+(j+32*x))];
            s[4096+(i+32*y)*64+(j+32*x)] = device_Dist[(blk_p+(i+32*y))*tn+(blk_j+(j+32*x))];
        }
    }

    __syncthreads();
    #pragma unroll 48
    for(int k = 0; k < 64; k++){
        val0 = min(val0, s[i*64+k]+s[4096+k*64+j]);
        val1 = min(val1, s[i*64+k]+s[4096+k*64+(j+32)]);
        val2 = min(val2, s[(i+32)*64+k]+s[4096+k*64+j]);
        val3 = min(val3, s[(i+32)*64+k]+s[4096+k*64+(j+32)]);
    }

    device_Dist[(blk_i+i)*tn+(blk_j+j)] = val0;
    device_Dist[(blk_i+i)*tn+(blk_j+(j+32))] = val1;
    device_Dist[(blk_i+(i+32))*tn+(blk_j+j)] = val2;
    device_Dist[(blk_i+(i+32))*tn+(blk_j+(j+32))] = val3;
}